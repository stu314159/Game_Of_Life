
#include <hip/hip_runtime.h>
#include <ctime>
#include <stdlib.h>
#include <iostream>

__device__ int sum_neighbors(int* board, int r, int c, int n) {
	int sum = 0;
	for(int i = r-1; i < r+2; i++) {
		for(int j = c-1; j < c+2; j++) {
			if((i != r) || (j != c)) {
				sum += board[i*n+j];
			}
		}
	}
	return sum;
}

__global__ void tick(int* board_in, int* board_out, int n) {
	
	//int my_index = (blockDim.x+2)*(blockIdx.x+1) + threadIdx.x+1;
	int row = blockIdx.x+1; //plus one to account for border remaining constant. blocks/threads index the inner matrix
	int col = threadIdx.x+1;
	if(board_in[row*n+col]){
		if(sum_neighbors(board_in, row, col, n)==2 || sum_neighbors(board_in, row, col, n)==3){
			board_out[row*n+col] = 1;
		}
		else{
			board_out[row*n+col] = 0;
		}
	}
	else{
		if(sum_neighbors(board_in, row, col, n)==3){
			board_out[row*n+col] = 1;
		}
		else{
			board_out[row*n+col] = 0;
		}
	}
}


int main(int argc, char* argv[]) {
	srand((unsigned) time(0));

	int n = atoi(argv[1]);
	int rounds = atoi(argv[2]);
	int* board_even = new int[n*n];
	int* board_odd = new int[n*n];
	

	//initialize random board
	for(int i = 0; i < n*n; i++) {
		board_even[i] = rand()%2;
		board_odd[i] = board_even[i];
	}
	//kill border, border stays dead
	for(int x = 0; x < n; x++) {
		board_even[x] = 0;
		board_even[(n-1)*n + x] = 0;
		board_even[x*n] = 0;
		board_even[x*n+n-1]=0;
		board_odd[x]=0;
		board_odd[(n-1)*n+x]=0;
		board_odd[x*n]=0;
		board_odd[x*n+n-1]=0;
	}

	int *board_even_d, *board_odd_d;
	hipMalloc(&board_even_d, n*n*sizeof(int));
	hipMalloc(&board_odd_d, n*n*sizeof(int));

	hipMemcpy(board_even_d, board_even, n*n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(board_odd_d, board_odd, n*n*sizeof(int), hipMemcpyHostToDevice);
	
	//check initial state, run 1 evolution, check end state
	
	for(int i = 0; i < n; i++){
                for(int j = 0; j < n; j++){
                        std::cout<<board_even[i*n+j]<<" ";
                }
                std::cout<<"\n";
        }
	std::cout<<"\n";

	for(int r = 0; r < rounds; r++) {
		//evolve
		if (r%2==0){
			tick<<<n-2, n-2>>>(board_even_d, board_odd_d, n);
		}
		else {
			tick<<<n-2, n-2>>>(board_odd_d, board_even_d, n);
		}
	}

	hipMemcpy(board_odd, board_odd_d, n*n*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(board_even, board_even_d, n*n*sizeof(int), hipMemcpyDeviceToHost);
	
	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++) {
			std::cout<<board_odd[i*n+j]<<" ";
		}
		std::cout<<"\n";
	}
	std::cout<<"\n";

	for(int i = 0; i < n; i++){
		for(int j = 0; j < n; j++){
			std::cout<<board_even[i*n+j]<<" ";
		}
		std::cout<<"\n";
	}

	hipFree(board_even_d);
	hipFree(board_odd_d);
	delete[] board_even;
	delete[] board_odd;
	return 0;
}